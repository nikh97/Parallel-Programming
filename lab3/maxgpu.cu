#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

unsigned int getmax(unsigned int *, unsigned int);

#define TPB 1024

__global__ void get_cuda_max(unsigned int* dev_num, unsigned int size){

  unsigned int id = (blockDim.x * blockIdx.x) + threadIdx.x;
  unsigned int size_cp = size;

  unsigned int ten = size_cp/10;

  if(id < ten){

    for(unsigned int i = 1; i < 10; i++){

      if(dev_num[ten*i + id] > dev_num[id])
        dev_num[id] = dev_num[ten*i + id];
    }
  }
}

int main(int argc, char *argv[])
{
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array
    
    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }
   
    size = atol(argv[1]);

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }    

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
       numbers[i] = rand()  % size;

    // for( i = 0; i < size; i++)
    //   printf("%u\n", numbers[i]);

    unsigned int num_blocks = (size + TPB - 1)/TPB;

    unsigned int* dev_num;

    hipMalloc((void**) &dev_num, size*sizeof(unsigned int));
    hipMemcpy(dev_num, numbers, size*sizeof(unsigned int), hipMemcpyHostToDevice);

    unsigned int size_cp = size;

    while(size_cp > 1){
      get_cuda_max<<<num_blocks, TPB>>>(dev_num, size_cp);
      size_cp = size_cp/10;
    }

    hipMemcpy(numbers, dev_num, size*sizeof(unsigned int), hipMemcpyDeviceToHost);

    unsigned int ans = numbers[0];

    hipFree(dev_num);
    
    printf(" The maximum number in the array is: %u\n", 
           ans);

    printf("The max num sequentially is: %u\n", getmax(numbers, size));

    free(numbers);

    exit(0);
}

/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/
unsigned int getmax(unsigned int num[], unsigned int size)
{
  unsigned int i;
  unsigned int max = num[0];

  for(i = 1; i < size; i++)
  if(num[i] > max)
     max = num[i];

  return( max );

}


